#include "hip/hip_runtime.h"
#pragma once

#include<hip/hip_runtime.h>
#include "constants.h"
#include "data_structures.h"

__global__ void recurrence1_no_const(DTYPE* a, DTYPE* b, DTYPE* c, int num_chunks, int n)
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx>=num_chunks)
        return;
    const unsigned int chunk_start = idx * n;

    for (int i = chunk_start + 1 ; i < chunk_start + n ; i++)
    {
        b[i] -= a[i]*c[i-1]/b[i-1];
    }

}

__global__ void recurrence1(DTYPE* a, DTYPE* b, DTYPE* c, int num_chunks)
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int n = M;
    if (idx>=num_chunks)
        return;
    const unsigned int chunk_start = idx * n;
    // const unsigned int chunk_end = chunk_start + n;
    
    DTYPE as[n-1];
    DTYPE bs[n];
    DTYPE cs[n-1];
    
    #pragma unroll
    for (int i = 0 ; i < n ; i++)
    {
        int loc = chunk_start + i;
        as[i] = a[loc+1];
        bs[i] = b[loc];  
        cs[i] = c[loc];
    }
    #pragma unroll
    for (int i = 0 ; i < n -1  ; i++)
    {
        bs[i+1] -= as[i]*cs[i]/bs[i];
    }
    #pragma unroll
    for (int i = 0 ; i < n ; i++)
    {
        b[chunk_start + i] = bs[i];
    }
}
__global__ 
void firstMap(DTYPE* a, DTYPE* b, DTYPE* c, tuple4<DTYPE>* tups, int total_size, int n)
{
    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx>=total_size)
        return;
    tuple4<DTYPE> t;
    if (idx % n == 0)
    {
        t.a = 1;
        t.b = 0;
        t.c = 0;
        t.d = 1;
    }
    else
    {
        t.a = b[idx];
        t.b = -a[idx] * c[idx-1];
        t.c = 1;
        t.d = 0;
    }
    tups[idx] = t;
}

__global__
void generate_keys(unsigned int* keys, int total_size, int n)
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx>=total_size)
        return;
    keys[idx] = idx / n;
}

__global__ void get_first_elem(DTYPE* in, DTYPE* out, int num_chunks, int n)
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx>=num_chunks)
        return;
    out[idx] = in[idx * n];
}

__global__ void map2(tuple4<DTYPE>* tups, unsigned int* keys, DTYPE* b, DTYPE* b0s, int total_size, int n)
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx>=total_size)
        return;
    tuple4<DTYPE> t = tups[idx];
    DTYPE b0 = b0s[keys[idx]];
    b[idx] = (t.a*b0 + t.b) / (t.c*b0 + t.d);
    
}

__global__
void map3(tuple2<DTYPE>* tups, DTYPE* a, DTYPE* b, DTYPE* d, int total_size, int n)
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx>=total_size)
        return;
    tuple2<DTYPE> t;
    if (idx % n == 0)
    {
        t.a = 0;
        t.b = 1;

    }
    else
    {
        t.a = d[idx];
        t.b = -a[idx]/b[idx-1];
    }
    tups[idx] = t;
}

__global__ void map4(tuple2<DTYPE>* tups, unsigned int* keys, DTYPE* d, DTYPE* d0s, int total_size, int n)
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx>=total_size)
        return;
    tuple2<DTYPE> t = tups[idx];
    DTYPE d0 = d0s[keys[idx]];
    d[idx] = t.a + t.b*d0;
}

__global__ void getLastDiv(DTYPE* d, DTYPE* b, DTYPE* lastDiv, int num_chunks, int n)
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx>=num_chunks)
        return;
    const int n1 = idx * n + (n-1);
    lastDiv[idx] = d[n1]/b[n1]; 
}

__global__ void map5(tuple2<DTYPE>* tups, unsigned int* keys, DTYPE* b, DTYPE* c, DTYPE* d, int total_size, int n)
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx>=total_size)
        return;
    const unsigned int revIdx = n * keys[idx] + (n - (idx%n) - 1);
    tuple2<DTYPE> t;
    if (idx % n == 0)
    {
        t.a = 0;
        t.b = 1;

    }
    else
    {
        t.a = d[revIdx]/b[revIdx];
        t.b = -c[revIdx]/b[revIdx];
    }
    tups[idx] = t;
}

__global__ void map6(tuple2<DTYPE>* tups, unsigned int* keys, DTYPE* lastDivs, DTYPE* d, int total_size, int n)
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx>=total_size)
        return;
    unsigned int k = keys[idx];
    const unsigned int revIdx = n * k + (n - (idx%n) - 1);
    tuple2<DTYPE> t = tups[idx];
    d[revIdx] =  t.a + t.b*lastDivs[k];
}


__global__
void execute_no_const(
    const DTYPE *a,
    const DTYPE *b,
    DTYPE *c,
    DTYPE *d,
    DTYPE *solution,
    int total_size,
    int n
){
    const size_t idx = (blockIdx.x * blockDim.x + threadIdx.x) * n;

    if (idx >= total_size) {
        return;
    }

    c[idx] /= b[idx];
    d[idx] /= b[idx];
    DTYPE norm_factor;
    #pragma unroll
    for (ptrdiff_t j = 1; j < n; ++j) {
        norm_factor = 1.0 / (b[idx+j] - a[idx+j] * c[idx + j-1]);
        c[idx + j] = c[idx+j] * norm_factor;
        d[idx + j] = (d[idx+j] - a[idx+j] * d[idx + j-1]) * norm_factor;
    }

    solution[idx + n-1] = d[idx + n-1];
    #pragma unroll
    for (ptrdiff_t j=n-2; j >= 0; --j) {
        solution[idx + j] = d[idx + j] - c[idx + j] * solution[idx + j+1];
    }
}

__global__
void execute(
    const DTYPE *a,
    const DTYPE *b,
    const DTYPE *c,
    const DTYPE *d,
    DTYPE *solution,
    int total_size
){
    // const int m = M;
    const size_t idx = (blockIdx.x * blockDim.x + threadIdx.x) * M;

    if (idx >= total_size) {
        return;
    }

    DTYPE cp[M];
    DTYPE dp[M];

    cp[0] = c[idx] / b[idx];
    dp[0] = d[idx] / b[idx];
    DTYPE norm_factor;
    #pragma unroll
    for (ptrdiff_t j = 1; j < M; ++j) {
        norm_factor = 1.0 / (b[idx+j] - a[idx+j] * cp[j-1]);
        cp[j] = c[idx+j] * norm_factor;
        dp[j] = (d[idx+j] - a[idx+j] * dp[j-1]) * norm_factor;
    }

    solution[idx + M-1] = dp[M-1];
    #pragma unroll
    for (ptrdiff_t j=M-2; j >= 0; --j) {
        solution[idx + j] = dp[j] - cp[j] * solution[idx + j+1];
    }
}